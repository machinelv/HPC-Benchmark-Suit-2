#include "hip/hip_runtime.h"
/*! @file ParabolicFunctionNC2Stage_GPU.cu
    @author Youngdae Kim
    @brief Evaluate the parabolic term using a 2-stage finite difference discretization.
*/

#include <basic_gpu.h>
#include <arrayfunctions_gpu.h>
#include <mpivars.h>
#include <hypar.h>

/*! Kernel for ParabolicFunctionNC2Stage_GPU() */
__global__
void ParabolicFunctionNC2Stage_kernel(
    int ngrid_points,
    int ghosts,
    int d1,
    int d2,
    int ndims,
    int nvars,
    const int *dim,
    const double *dxinv,
    const double *Deriv2,
    double *par
)
{
    int tx = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tx < ngrid_points) {
        int p, v;
        int index[GPU_MAX_NDIMS];
        double dxinv1, dxinv2;

        _ArrayIndexnD_(ndims,tx,dim,index,0);
        _ArrayIndex1D_(ndims,dim,index,ghosts,p);
        _GetCoordinate_(d1,index[d1],dim,ghosts,dxinv,dxinv1);
        _GetCoordinate_(d2,index[d2],dim,ghosts,dxinv,dxinv2);
        for (v=0; v<nvars; v++) par[nvars*p+v] += (dxinv1*dxinv2 * Deriv2[nvars*p+v]);
    }

    return;
}

/*! Evaluate the parabolic term using a "1.5"-stage finite-difference spatial discretization:
    The parabolic term is assumed to be of the form:
    \f{equation}{
      {\bf P}\left({\bf u}\right) = \sum_{d1=0}^{D-1}\sum_{d2=0}^{D-1} \frac {\partial^2 h_{d1,d2}\left(\bf u\right)} {\partial x_{d1} \partial x_{d2}},
    \f}
    where \f$d1\f$ and \f$d2\f$ are spatial dimension indices, and \f$D\f$ is the total number of spatial dimensions (#HyPar::ndims). This term is
    discretized at a grid point as:
    \f{equation}{
      \left.{\bf P}\left({\bf u}\right)\right|_j = \sum_{d1=0}^{D-1} \sum_{d2=0}^{D-1} \frac { \mathcal{D}_{d1}\mathcal{D}_{d2} \left[ {\bf h}_{d1,d2} \right] } {\Delta x_{d1} \Delta x_{d2}},
    \f}
    where \f$\mathcal{D}\f$ denotes the finite-difference approximation to the first derivative. Each of the first derivative approximations are \f$\mathcal{D}_{d1}\f$ and \f$\mathcal{D}_{d2}\f$ are computed separately, and thus the cross-derivative is evaluated in two steps using #HyPar::FirstDerivativePar.

    \b Notes:
    + This form of the parabolic term \b does \b allow for cross-derivatives (\f$ d1 \ne d2 \f$).
    + A \f$n\f$-th order central approximation to the second derivative can be expressed as a
      conjugation of two \f$(n-1)\f$-th order approximations to the first
      derivative, one forward and one backward. Computing it this way avoids
      odd-even decoupling. Thus, where possible #HyPar::FirstDerivativePar should
      point to the function computing \f$(n-1)\f$-th order first derivative where \f$n\f$
      is the desired order. Currently, this is implemented only for \f$n=2\f$. For other values
      of \f$n\f$, the first derivative is also computed with a \f$n\f$-th order approximation.

    To use this form of the parabolic term:
    + specify \b "par_space_type" in solver.inp as \b "nonconservative-2stage" (#HyPar::spatial_type_par).
    + the physical model must specify \f${\bf h}_{d1,d2}\left({\bf u}\right)\f$ through #HyPar::HFunction.
*/
int ParabolicFunctionNC2Stage_GPU(
    double  *par, /*!< array to hold the computed parabolic term */
    double  *u,   /*!< solution */
    void    *s,   /*!< Solver object of type #HyPar */
    void    *m,   /*!< MPI object of type #MPIVariables */
    double  t     /*!< Current simulation time */
)
{
  HyPar         *solver = (HyPar*)        s;
  MPIVariables  *mpi    = (MPIVariables*) m;
  double        *Func   = solver->fluxC;
  double        *Deriv1 = solver->Deriv1;
  double        *Deriv2 = solver->Deriv2;
  int           d1, d2;
  _DECLARE_IERR_;

  int     ndims  = solver->ndims;
  int     nvars  = solver->nvars;
  int     ghosts = solver->ghosts;
  int     *dim   = solver->dim_local;
  int     size   = solver->npoints_local_wghosts;

  if (!solver->HFunction) return(0); /* zero parabolic terms */
  solver->count_par++;

  gpuArraySetValue(par, size*nvars, 0.0);

  int ngrid_points = 1; for (int i = 0; i < ndims; i++) ngrid_points *= dim[i];

  for (d1 = 0; d1 < ndims; d1++) {
    for (d2 = 0; d2 < ndims; d2++) {

      /* calculate the diffusion function */
      solver->HFunction(Func,u,d1,d2,solver,t);
      solver->FirstDerivativePar(Deriv1,Func  ,d1, 1,solver,mpi);
      MPIExchangeBoundariesnD(ndims,nvars,dim,ghosts,mpi,Deriv1);
      solver->FirstDerivativePar(Deriv2,Deriv1,d2,-1,solver,mpi);

      /* calculate the final term - second derivative of the diffusion function */
      int nblocks = (ngrid_points - 1) / GPU_THREADS_PER_BLOCK + 1;
      ParabolicFunctionNC2Stage_kernel<<<nblocks, GPU_THREADS_PER_BLOCK>>>(
          ghosts, d1, d2, ndims, nvars, ngrid_points,
          solver->gpu_dim_local, solver->gpu_dxinv, solver->Deriv2, par
      );

      /*
      done = 0; _ArraySetValue_(index,ndims,0);
      while (!done) {
        _ArrayIndex1D_(ndims,dim,index,ghosts,p);
        _GetCoordinate_(d1,index[d1],dim,ghosts,dxinv,dxinv1);
        _GetCoordinate_(d2,index[d2],dim,ghosts,dxinv,dxinv2);
        for (v=0; v<nvars; v++) par[nvars*p+v] += (dxinv1*dxinv2 * Deriv2[nvars*p+v]);
        _ArrayIncrementIndex_(ndims,dim,index,done);
      }
      */
    }
  }

  if (solver->flag_ib) _ArrayBlockMultiply_(par,solver->iblank,size,nvars);
  return(0);
}
